#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;


//////////////////////////////////////////////////////////////////////////////////////////
__global__ void computeCost(const double *Params, const float *uproj, const float *mu, const float *W, 
        const int *ioff, const bool *iW, float *cmax){
    
  int tid, bid, Nspikes, Nfeatures, NfeatW, Nthreads, k;
  float xsum = 0.0f, Ci, lam; 
  
  Nspikes               = (int) Params[0];
  Nfeatures             = (int) Params[1];
  NfeatW                = (int) Params[4];
  Nthreads              = blockDim.x;
  lam                   = (float) Params[5];
    
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  while(tid<Nspikes){
      if (iW[tid + bid*Nspikes]){
          xsum = 0.0f;
          for (k=0;k<Nfeatures;k++)
              xsum += uproj[k + Nfeatures * tid] * W[k + ioff[tid] +  NfeatW * bid];
          
          Ci = max(0.0f, xsum) + lam/mu[bid];
          
          cmax[tid + bid*Nspikes] = Ci * Ci / (1.0f + lam/(mu[bid] * mu[bid])) - lam;          
      }
      tid+= Nthreads;
  }
  
}


//////////////////////////////////////////////////////////////////////////////////////////
__global__ void bestFilter(const double *Params, const bool *iW, const float *cmax, int *id){
    
  int tid,tind,bid, ind, Nspikes, Nfilters, Nthreads, Nblocks;
  float max_running = 0.0f, Th; 
  
  Nspikes               = (int) Params[0];
  Nfilters              = (int) Params[2];
  Nthreads              = blockDim.x;
  Nblocks               = gridDim.x;
  Th                    = (float) Params[7];

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  tind = tid + bid * Nthreads;
  
  while (tind<Nspikes){
      max_running = 0.0f;
      id[tind] = 0;
      
      for(ind=0; ind<Nfilters; ind++)
          if (iW[tind + ind*Nspikes])
              if (cmax[tind + ind*Nspikes] > max_running){
                  id[tind] = ind;
                  max_running = cmax[tind + ind*Nspikes];
              }

        if (max_running < Th*Th)
            id[tind] = -1;
              
      tind += Nblocks*Nthreads; 
  }  
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void average_snips(const double *Params, const int *ioff, const int *id, const float *uproj, 
        const float *cmax, const int *iList, float *cf, float *WU){
    
  int tid, bid, ind, Nspikes, Nfeatures, NfeatW, Nnearest, t;
  float xsum = 0.0f, pm; 
  
  Nspikes               = (int) Params[0];
  Nfeatures             = (int) Params[1];
  pm                    = (float) Params[3];
  NfeatW                = (int) Params[4];
  Nnearest              = (int) Params[6];
 
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  for(ind=0; ind<Nspikes;ind++)
      if (id[ind]==bid){
          
          xsum = uproj[tid + Nfeatures * ind];
          WU[tid + ioff[ind] + NfeatW * bid] = pm * WU[tid + ioff[ind] + NfeatW * bid] 
                  + (1-pm) * xsum;
          
          // go through the top 10 nearest filters and match them 
          for (t=0;t<Nnearest;t++)
              cf[ind + t*Nspikes] = cmax[ind + Nspikes * iList[t + Nnearest*bid]];
          
      }  
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void count_spikes(const double *Params, const int *id, int *nsp){
    
  int tid, tind, bid, ind, Nspikes, Nfilters, Nthreads, Nblocks;
  
  Nspikes               = (int) Params[0];
  Nfilters             = (int) Params[2];
  
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
   Nthreads              = blockDim.x;
  Nblocks               = gridDim.x;
  
  tind = tid + Nthreads *bid;
  
  while (tind<Nfilters){
      for(ind=0; ind<Nspikes;ind++)
          if (id[ind]==tind)
              nsp[tind] += 1;
      tind += Nthreads * Nblocks;
  }
}


//////////////////////////////////////////////////////////////////////////////////////////

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare input variables*/
  double *Params, *d_Params;
  int Nfeatures, Nspikes, Nfilters, Nnearest;
  
  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  /* read Params and copy to GPU */
  Params                = (double*) mxGetData(prhs[0]);
  Nspikes               = (int) Params[0];
  Nfeatures             = (int) Params[1];
  Nfilters              = (int) Params[2];
  Nnearest              = (int) Params[6];
  
  // copy Params to GPU
  hipMalloc(&d_Params,      sizeof(double)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(double)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);
  
  /* collect input GPU variables*/
  mxGPUArray const  *W, *uproj, *ioff, *iW, *mu, *iList;
  const float *d_W, *d_uproj, *d_mu;
  const int *d_ioff, *d_iList;
  const bool *d_iW;
  float *d_dWU;
    
  // these come as const GPU Arrays, just transfer them over
  uproj         = mxGPUCreateFromMxArray(prhs[1]);
  W             = mxGPUCreateFromMxArray(prhs[2]);
  ioff          = mxGPUCreateFromMxArray(prhs[3]);  
  iW            = mxGPUCreateFromMxArray(prhs[4]);
  mu            = mxGPUCreateFromMxArray(prhs[6]);
  iList         = mxGPUCreateFromMxArray(prhs[7]);

  d_uproj       = (float const *)(mxGPUGetDataReadOnly(uproj));
  d_W        	= (float const *)(mxGPUGetDataReadOnly(W));
  d_ioff        = (int const *)  (mxGPUGetDataReadOnly(ioff));
    // this has a one for filter - spike combinations to be considered
  d_iW          = (bool const *)  (mxGPUGetDataReadOnly(iW));
  d_mu          = (float const *)  (mxGPUGetDataReadOnly(mu));
  d_iList       = (int const *)  (mxGPUGetDataReadOnly(iList));

  // dWU is not a constant , so the data has to be "copied" over
  mxGPUArray *dWU;
  dWU       = mxGPUCopyFromMxArray(prhs[5]);
  d_dWU     = (float *)(mxGPUGetData(dWU));  
  
  /* Define new GPU variables*/
  float *d_cmax, *d_cf;
  int *d_id, *d_nsp;
  
  // allocate a lot of GPU variables
  hipMalloc(&d_cmax,    Nspikes * Nfilters *  sizeof(float));
  hipMalloc(&d_id,      Nspikes  *  sizeof(int));
  hipMalloc(&d_nsp,      Nfilters  *  sizeof(int));
  hipMalloc(&d_cf,      Nspikes  * Nnearest * sizeof(float));
   
  hipMemset(d_nsp,      0, Nfilters *   sizeof(int));
  
  // get list of cmaxes for each combination of neuron and filter
  computeCost<<<Nfilters, 1024>>>(d_Params, d_uproj, d_mu, d_W, d_ioff, 
          d_iW, d_cmax);

  // loop through cmax to find best template
  bestFilter<<<40, 256>>>(d_Params, d_iW, d_cmax, d_id);
  
  // average all spikes for same template
  average_snips<<<Nfilters, Nfeatures>>>(d_Params, d_ioff, d_id, d_uproj, 
          d_cmax, d_iList, d_cf,  d_dWU);
  
  count_spikes<<<7, 256>>>(d_Params, d_id, d_nsp);

  // dWU stays a GPU array
  plhs[0] 	= mxGPUCreateMxArrayOnGPU(dWU);
  
  // put these ones on the CPU side: id, cmax, cf, nsp 
  int *id, *nsp;
  float *cmax, *cf;
  
  const mwSize dimst[] 	= {Nspikes,1};  
  const mwSize dimst2[] 	= {Nspikes,Nfilters};  
  const mwSize dimst3[] 	= {Nspikes,Nnearest};  
  const mwSize dimst4[] 	= {Nfilters,1};  

  plhs[1]   = mxCreateNumericArray(2, dimst,  mxINT32_CLASS,  mxREAL);
  plhs[2]   = mxCreateNumericArray(2, dimst2, mxSINGLE_CLASS, mxREAL);
  plhs[3]   = mxCreateNumericArray(2, dimst3, mxSINGLE_CLASS, mxREAL);
  plhs[4]   = mxCreateNumericArray(2, dimst4, mxINT32_CLASS,  mxREAL);

  id        = (int*) mxGetData(plhs[1]);  
  cmax      = (float*) mxGetData(plhs[2]);  
  cf        = (float*) mxGetData(plhs[3]);  
  nsp       = (int*) mxGetData(plhs[4]);  
  
  hipMemcpy(id,   d_id,  Nspikes * sizeof(int),   hipMemcpyDeviceToHost);
  hipMemcpy(cmax, d_cmax,Nspikes * Nfilters* sizeof(float),  hipMemcpyDeviceToHost);
  hipMemcpy(cf,   d_cf,  Nspikes * Nnearest* sizeof(float),  hipMemcpyDeviceToHost);
  hipMemcpy(nsp,  d_nsp, Nfilters * sizeof(int),   hipMemcpyDeviceToHost);
  
  //we are done, clear everything from the GPU
  hipFree(d_Params);
  hipFree(d_cmax);
  hipFree(d_id);
  hipFree(d_cf);

  //do this for the constant variables
  mxGPUDestroyGPUArray(uproj);
  mxGPUDestroyGPUArray(dWU);  
  mxGPUDestroyGPUArray(W);    
  mxGPUDestroyGPUArray(ioff);  
  mxGPUDestroyGPUArray(iW);  
  mxGPUDestroyGPUArray(mu);  
  mxGPUDestroyGPUArray(iList);  

  
}
