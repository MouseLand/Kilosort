#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;

const int nblock = 32;
//////////////////////////////////////////////////////////////////////////////////////////

__global__ void	crossFilter(const double *Params, const float *W1, const float *W2,
        const float *UtU, float *WtW){    
  __shared__ float shW1[nblock*81], shW2[nblock*81]; 

  float x;
  int nt0, tidx, tidy , bidx, bidy, i, Nfilt, t, tid1, tid2;

  tidx 		= threadIdx.x;
  tidy 		= threadIdx.y;
  bidx 		= blockIdx.x;
  bidy 		= blockIdx.y;
  
  Nfilt     = (int) Params[1];
  nt0       = (int) Params[9];
  
  tid1 = tidx + bidx*nblock;
  
  tid2 = tidy + bidx*nblock;
  if (tid2<Nfilt){
      while(tidx<nt0){
          shW1[tidx + tidy * nt0] = W1[tidx + tid2 * nt0];
          tidx+= nblock;
      }
  }
  tidx 		= threadIdx.x;
  tid2      = tidy + bidy*nblock;
  if (tid2<Nfilt){
      while(tidx<nt0){
          shW2[tidx + tidy * nt0] = W2[tidx + tid2 * nt0];
          tidx+= nblock;
      }
  }
  tidx 		= threadIdx.x;

  __syncthreads();
      
  if (tid2<Nfilt && tid1<Nfilt){
      for(i=0;i<2*nt0-1;i++){
          x = 0.0f;
          if(i<nt0)
              for(t=0;t<i+1;t++)
                  x += shW1[t + nt0 * tidx] * shW2[t + (nt0-i-1) + nt0 * tidy];
          else
              for(t=i-nt0+1;t<nt0;t++)
                  x += shW1[t + nt0 * tidx] * shW2[t + (nt0-i-1) + nt0 * tidy];
          
          WtW[tid1 + tid2*Nfilt +  i*Nfilt*Nfilt] =
                  x * UtU[tid1 + tid2*Nfilt];
      }
  }
}


//////////////////////////////////////////////////////////////////////////////////////////

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare input variables*/
  double *Params, *d_Params;
  unsigned int nt0, Nfilt;

  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  /* read Params and copy to GPU */
  Params  	= (double*) mxGetData(prhs[0]);
  Nfilt		= (unsigned int) Params[1];
  nt0       = (unsigned int) Params[9];
  
  hipMalloc(&d_Params,      sizeof(double)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(double)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);

  /* collect input GPU variables*/
  mxGPUArray const  *W1, *W2,   *UtU;
  const float     *d_W1,*d_W2, *d_UtU;
  
  W1             = mxGPUCreateFromMxArray(prhs[1]);
  d_W1        	= (float const *)(mxGPUGetDataReadOnly(W1));
  W2             = mxGPUCreateFromMxArray(prhs[2]);
  d_W2        	= (float const *)(mxGPUGetDataReadOnly(W2));
  UtU       	= mxGPUCreateFromMxArray(prhs[3]);
  d_UtU     	= (float const *)(mxGPUGetDataReadOnly(UtU));


  mxGPUArray *WtW;
  float  *d_WtW;
  const mwSize dimsu[] 	= {Nfilt, Nfilt, 2*nt0-1}; 
  WtW 		= mxGPUCreateGPUArray(3, dimsu, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
  d_WtW 		= (float *)(mxGPUGetData(WtW));

  dim3 grid(1 + (Nfilt/nblock), 1 + (Nfilt/nblock));
  dim3 block(nblock, nblock);
  crossFilter<<<grid, block>>>(d_Params, d_W1, d_W2, d_UtU, d_WtW); 

  plhs[0] 	= mxGPUCreateMxArrayOnGPU(WtW);

  hipFree(d_Params);
  mxGPUDestroyGPUArray(WtW);
  mxGPUDestroyGPUArray(W1);
  mxGPUDestroyGPUArray(W2);
  mxGPUDestroyGPUArray(UtU);
  
}
