#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;

//for sorting according to timestamps
//#include "mexNvidia_quicksort.cu"



const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	spaceFilter(const double *Params, const float *data, const float *U, 
        const int *iC, const int *iW, float *dprod){    
  volatile __shared__ float  sU[32*NrankMax];
  volatile __shared__ int iU[32]; 
  float x;
  int tid, bid, i,k, Nrank, Nchan, NT, Nfilt, NchanU;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT      	=   (int) Params[0];
  Nfilt    	=   (int) Params[1];
  Nrank     = (int) Params[6];
  NchanU    = (int) Params[10];
  Nchan     = (int) Params[9];
  
  if (tid<NchanU)
      iU[tid] = iC[tid + NchanU * iW[bid]];
  __syncthreads();  
  
  if(tid<NchanU*Nrank)
      sU[tid]= U[iU[tid%NchanU] + Nchan * bid + Nchan * Nfilt * (tid/NchanU)];
        
  //sU[tid]= U[tid%NchanU + NchanU * bid + NchanU * Nfilt * (tid/NchanU)];
  
  __syncthreads();  
  
  while (tid<NT){
      for (k=0;k<Nrank;k++){
          x = 0.0f;
          for(i=0;i<NchanU;i++)
              x  += sU[i + NchanU*k] * data[tid + NT * iU[i]];
          dprod[tid + NT*bid + k*NT*Nfilt]   = x;
      }
      
      tid += blockDim.x;
      __syncthreads();
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	spaceFilterUpdate(const double *Params, const float *data, const float *U, const bool *UtU,
        const int *iC, const int *iW, float *dprod,  const int *st, const int *id, const int *counter){
    volatile __shared__ float  sU[32*NrankMax];
    volatile __shared__ int iU[32];
    float x;
    int tid, bid, ind, nt0, i, t, k, Nrank, NT, Nfilt, NchanU, Nchan;
    
    tid 		= threadIdx.x;
    bid 		= blockIdx.x;
    NT      	= (int) Params[0];
    Nfilt    	= (int) Params[1];
    Nrank     = (int) Params[6];
    NchanU    = (int) Params[10];
    nt0       = (int) Params[4];
    Nchan     = (int) Params[9];
    
    // just need to do this for all filters that have overlap with id[bid] and st[id]
    // tidx still represents time, from -nt0 to nt0
    // tidy loops through all filters that have overlap
    
    if (tid<NchanU)
        iU[tid] = iC[tid + NchanU * iW[bid]];
    __syncthreads();
    
    if (tid<NchanU)
       for (k=0;k<Nrank;k++)
            sU[tid + k * NchanU] = U[iU[tid] + Nchan * bid + Nchan * Nfilt * k];

    __syncthreads();
    
    for(ind=counter[1];ind<counter[0];ind++)
        if (UtU[id[ind] + Nfilt *bid]){
            t = st[ind] + tid - nt0;
            // if this is a hit, threads compute all time offsets
            if (t>=0 & t<NT){
                for (k=0;k<Nrank;k++){
                    x = 0.0f;
                    for(i=0;i<NchanU;i++)
                        x  += sU[i + NchanU*k] * data[t + NT * iU[i]];
                    dprod[t + NT*bid + k*NT*Nfilt]   = x;
                }
            }            
        }
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	timeFilter(const double *Params, const float *data, const float *W,float *conv_sig){    
  volatile __shared__ float  sW2[81*NrankMax], sW[81*NrankMax], sdata[(Nthreads+81)*NrankMax]; 
  float x;
  int tid, tid0, bid, i, nid, Nrank, NT, Nfilt, nt0;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT      	=   (int) Params[0];
  Nfilt    	=   (int) Params[1];
  Nrank     = (int) Params[6];
  nt0       = (int) Params[4];
  
  if(tid<nt0*Nrank)
      sW[tid]  = W[tid%nt0 + (bid + Nfilt * (tid/nt0))* nt0];

  __syncthreads();
  
  tid0 = 0;
  while (tid0<NT-Nthreads-nt0+1){
	  if (tid<nt0*NrankMax) 
          sdata[tid%nt0 + (tid/nt0)*(Nthreads+nt0)] = 
			data[tid0 + tid%nt0+ NT*(bid + Nfilt*(tid/nt0))];
	  
      #pragma unroll 3
      for(nid=0;nid<Nrank;nid++){
          sdata[tid + nt0+nid*(Nthreads+nt0)] = data[nt0+tid0 + tid+ NT*(bid +nid*Nfilt)];
	  }
	  __syncthreads();
      
	  x = 0.0f;      
      for(nid=0;nid<Nrank;nid++){
		  #pragma unroll 4
          for(i=0;i<nt0;i++)
              x    += sW[i + nid*nt0]  * sdata[i+tid + nid*(Nthreads+nt0)];                        
	  }
      conv_sig[tid0  + tid + NT*bid]              = x;      
      
      tid0+=Nthreads;
      __syncthreads();
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	timeFilterUpdate(const double *Params, const float *data, const float *W,
        const bool *UtU, float *conv_sig, const int *st, const int *id, const int *counter){    
    
  volatile __shared__ float  sW[81*NrankMax], sW2[81*NrankMax]; 
  float x;
  int tid, tid0, bid, t, k,ind, Nrank, NT, Nfilt, nt0;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT      	=   (int) Params[0];
  Nfilt    	=   (int) Params[1];
  Nrank     = (int) Params[6];
  nt0       = (int) Params[4];
  
   if (tid<nt0)
       for (k=0;k<Nrank;k++)
           sW[tid + k*nt0]= W[tid + nt0*bid + nt0*Nfilt * k];
  __syncthreads();
  
  for(ind=counter[1];ind<counter[0];ind++)
      if (UtU[id[ind] + Nfilt *bid]){
          tid0 = st[ind] - nt0 + tid;
          if (tid0>=0 && tid0<NT-nt0){
              x = 0.0f;
              for (k=0;k<Nrank;k++)
                  for (t=0;t<nt0;t++)
                      x += sW[t +k*nt0] * data[t + tid0 + NT * bid + NT * Nfilt *k];                  
       
              conv_sig[tid0 + NT*bid]   = x;
          }
          
      }
  
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void  bestFilter(const double *Params, const float *data, 
	const float *mu, float *err, float *eloss, int *ftype){
  int tid, tid0, i, bid, NT, Nfilt, ibest = 0, nt0;
  float  Cf, Cbest, lam, b, a, Cnextbest;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT 		= (int) Params[0];
  Nfilt 	= (int) Params[1];
  lam 	    = (float) Params[7];
  nt0       = (int) Params[4];
  
  tid0 = tid + bid * blockDim.x;
  while (tid0<NT-nt0){
      Cbest = 0.0f;
      Cnextbest = 0.0f;
      
      for (i=0; i<Nfilt;i++){

          a = 1+ lam;
          b = max(0.0f, data[tid0 + NT * i]) + lam * mu[i];
          Cf =  b*b/a - lam * mu[i]*mu[i];
                  
          if (Cf > Cbest + 1e-6){
              Cnextbest = Cbest;
              Cbest 	= Cf;
              ibest 	= i;
          }
          else 
              if  (Cf > Cnextbest + 1e-6)
                    Cnextbest = Cf;                
      }
      err[tid0] 	= Cbest;
      eloss[tid0] 	= Cbest - Cnextbest;
      ftype[tid0] 	= ibest;
      
      tid0 += blockDim.x * gridDim.x;
  }
}

// THIS UPDATE DOES NOT UPDATE ELOSS?
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void  bestFilterUpdate(const double *Params, const float *data, 
	const float *mu, float *err, float *eloss, int *ftype, const int *st, const int *id, const int *counter){
  int tid,  ind, i,t, NT, Nfilt, ibest = 0, nt0;
  float  Cf, Cbest, lam, b, a, Cnextbest;

  tid 		= threadIdx.x;  
  NT 		= (int) Params[0];
  Nfilt 	= (int) Params[1];
  lam 	    = (float) Params[7];
  nt0       = (int) Params[4];
  
  
  // we only need to compute this at updated locations
  ind = counter[1] + blockIdx.x;
  
  if (ind<counter[0]){
      t = st[ind]-nt0 + tid;
      if (t>=0 && t<NT){
          Cbest = 0.0f;
          for (i=0; i<Nfilt;i++){
              a = 1+ lam;
              b = max(0.0f, data[t + NT * i]) + lam * mu[i];
              
              Cf =  b*b/a - lam * mu[i]*mu[i];

               if (Cf > Cbest + 1e-6){
                  Cnextbest = Cbest;
                  Cbest 	= Cf;
                  ibest 	= i;
              }
              else
                  if  (Cf > Cnextbest + 1e-6)
                      Cnextbest = Cf;
          }
          err[t] 	= Cbest;
          ftype[t] 	= ibest;
      }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	cleanup_spikes(const double *Params, const float *data, 
        const float *mu, const float *err, const float *eloss, const int *ftype, int *st, 
        int *id, float *x, float *y,  float *z, int *counter){
    
  int lockout, indx, tid, bid, NT, tid0,  j, id0, t0;
  volatile __shared__ float sdata[Nthreads+2*81+1];
  bool flag=0;
  float err0, Th;
  
  lockout   = (int) Params[4] - 1;
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  NT      	=   (int) Params[0];
  tid0 		= bid * blockDim.x ;
  Th 		= (float) Params[2];
  //lam 	    = (float) Params[7];
  
  while(tid0<NT-Nthreads-lockout+1){
      if (tid<2*lockout)
          sdata[tid] = err[tid0 + tid];
      sdata[tid+2*lockout] = err[2*lockout + tid0 + tid];
      
      __syncthreads();
      
      err0 = sdata[tid+lockout];
      if(err0>Th*Th){
          flag = 0;
          for(j=-lockout;j<=lockout;j++)
              if(sdata[tid+lockout+j]>err0){
                  flag = 1;
                  break;
              }
          if(flag==0){
              indx = atomicAdd(&counter[0], 1);
              if (indx<maxFR){
                  t0        = tid+lockout+tid0;
                  id0       = ftype[t0];                  
                  st[indx] = t0;
                  id[indx] = id0;
                  y[indx]  = data[t0 + NT * id0];
                  
                  //a = 1+ lam;
                  //b = max(0.0f, data[t0 + NT * id0]) + lam * mu[id0];
                  
                  x[indx] = sqrt(err0);
                  //x[indx]  = b/a;    // do I really need this here?
                  //x[indx]  = y[indx]; 
                  z[indx]  = eloss[t0];   
              }
          }
      }
      
      tid0 += blockDim.x * gridDim.x;
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	extractFEAT(const double *Params, const int *st, const int *id,
        const int *counter, const float *dout, const int *iList,
        const float *mu, float *d_feat){
    int t, tidx, tidy,Nblocks,NthreadsX,idF, bid,  NT, ind, tcurr, Nnearest;
    float rMax, Ci, Cf, lam;
    tidx 		= threadIdx.x;
    tidy 		= threadIdx.y;
    
    bid 		= blockIdx.x;
    NT 		= (int) Params[0];
    Nnearest 	= (int) Params[5];
    NthreadsX 	= blockDim.x;
    Nblocks               = gridDim.x;
    lam 	    = (float) Params[7];
    
    // each thread x does a nearby filter
    // each thread x combines with blocks to go through all new spikes    
    ind = counter[1]+tidx + NthreadsX * bid;
    
    while(ind<counter[0]){
        tcurr = st[ind];
        rMax = 0.0f;
        idF = iList[tidy + Nnearest * id[ind]];
        
        for (t=-3;t<3;t++){
            Ci = dout[tcurr +t+ idF * NT] + lam/mu[idF];
            Cf = Ci / sqrt(lam/(mu[idF] * mu[idF]) + 1.0f);
            rMax = max(rMax, Cf);
        }
        d_feat[tidy + ind * Nnearest] = rMax;
        ind += NthreadsX * Nblocks;
    }
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	subtract_spikes(const double *Params,  const int *st, 
        const int *id, const float *x, const int *counter, float *dataraw, 
        const float *W, const float *U){
  int nt0, tidx, tidy, k, NT, ind, Nchan, Nfilt, Nrank;
  float X;

  NT        = (int) Params[0];
  nt0       = (int) Params[4];
  Nchan     = (int) Params[9];
  Nfilt    	=   (int) Params[1];
  Nrank     = (int) Params[6];
  
  tidx 		= threadIdx.x;
  ind       = counter[1]+blockIdx.x;
  
  while(ind<counter[0]){
      tidy = threadIdx.y;
      
      while (tidy<Nchan){
          X = 0.0f;          
          for (k=0;k<Nrank;k++)
              X += W[tidx + id[ind]* nt0 + nt0*Nfilt*k] * 
                      U[tidy + id[ind] * Nchan + Nchan*Nfilt*k];                        
          
          dataraw[tidx + st[ind] + NT * tidy] -= x[ind] * X;          
          tidy += blockDim.y;
      }
      ind += gridDim.x;
  }
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void average_snips(const double *Params, const int *st, 
        const int *id,  const float *x, const float *y,  const int *counter, const float *dataraw, 
        const float *W, const float *U, double *WU, int *nsp, 
        const float *mu, const float *z){
    
  int nt0, tidx, tidy, bid, NT, Nchan,k, Nrank, Nfilt;
  int currInd;  
  float Th;
  double  X, xsum;
  
  NT        = (int) Params[0];
  Nfilt    	=   (int) Params[1];
  nt0       = (int) Params[4];
  Nrank     = (int) Params[6];
  Nchan     = (int) Params[9];  
  
  tidx 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  //Th = 10.f;
  Th 		= (float) Params[15];
  
  // we need wPCA projections in here, and then to decide based on total
  
  // idx is the time sort order of the spikes; the original order is a function
  // of when threads complete in mexGetSpikes. Compilation of the sums for WU, sig, and dnextbest
  // in a fixed order makes the calculation deterministic.
  
  for(currInd=0; currInd<counter[0];currInd++) {
      // only do this if the spike is "GOOD"
      if (x[currInd]>Th){
          if (id[currInd]==bid){
              if (tidx==0 &&  threadIdx.y==0) 
                  nsp[bid]++;                  
              
              tidy 		= threadIdx.y;              
              while (tidy<Nchan){
                  X = 0.0f;
                  for (k=0;k<Nrank;k++)
                      X += W[tidx + bid* nt0 + nt0*Nfilt*k] *
                              U[tidy + bid * Nchan + Nchan*Nfilt*k];
                  
                  xsum = dataraw[st[currInd]+tidx + NT * tidy] + y[currInd] * X;
                  
                  //WU[tidx+tidy*nt0 + nt0*Nchan * bid] *= p[bid];
                  WU[tidx+tidy*nt0 + nt0*Nchan * bid] += (double) xsum;
                  
                  tidy+=blockDim.y;
                  
              }        //end of while loop over channels
          }               //end of if block for id == bid
      }
  }                  //end of for loop over spike indicies
}                      //end of function






//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	computePCfeatures(const double *Params, const int *counter,
        const float *dataraw,  const int *st, const int *id, const float *x,
        const float *W, const float *U, const float *mu, const int *iW, const int *iC,
        const float *wPCA, float *featPC){
    
  volatile __shared__ float  sPCA[81 * NrankMax], sW[81 * NrankMax], sU[NchanMax * NrankMax];
  volatile __shared__ int iU[NchanMax];

  int bid, nt0, t, tidx, tidy, k, NT, ind, Nchan, NchanU, Nfilt, Nrank;
  float X = 0.0f, Y = 0.0f;

  NT        = (int) Params[0];
  nt0       = (int) Params[4];
  Nchan     = (int) Params[9];
  Nfilt    	= (int) Params[1];
  Nrank     = (int) Params[6];  
  NchanU    = (int) Params[10];
  
  tidx 		= threadIdx.x;
  tidy 		= threadIdx.y;
  bid       = blockIdx.x;
  
  if (tidy==0)
      iU[tidx] = iC[tidx + NchanU * iW[bid]];
  __syncthreads();

  sU[tidx + tidy*NchanU]= U[iU[tidx] + Nchan * bid + Nchan * Nfilt * tidy];
  
  while (tidx<nt0){
     sW[tidx + tidy*nt0]  = W[tidx + bid*nt0 + Nfilt * nt0 * tidy];
      sPCA[tidx + tidy*nt0]  = wPCA[tidx + nt0 * tidy];
      tidx += blockDim.x;
  }
  
  tidx 		= threadIdx.x;
  __syncthreads();
    
//   first, compute wPCA projections of the filter  
  Y = 0.0f;
  for (k =0; k<Nrank; k++){
      X = 0.0f;
      for (t=0;t<nt0;t++)
          X += sW[t + k*nt0] * sPCA[t + tidy * nt0];
      Y += X * sU[tidx + k*NchanU];
  }
  
  //now for each matching spike, compute the features
  for(ind=0; ind<counter[0];ind++)
      if (id[ind]==bid){          
          X = Y * x[ind]; // - mu[bid]); 
          for (t=0;t<nt0; t++)
              X  += dataraw[st[ind] + t + NT * iU[tidx]] * sPCA[t + nt0*tidy];
          featPC[tidx + tidy*NchanU + ind * NchanU*Nrank] = X;
      }
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	addback_spikes(const double *Params,  const int *st, 
        const int *id, const float *x, const int *count, float *dataraw, 
        const float *W, const float *U, const int iter, const float *spkscore){
  int nt0, tidx, tidy, k, NT, ind, Nchan, Nfilt, Nrank;
  float X, ThS;

  NT        = (int) Params[0];
  nt0       = (int) Params[4];
  Nchan     = (int) Params[9];
  Nfilt    	=   (int) Params[1];
  Nrank     = (int) Params[6];
  ThS      = (float) Params[11];
  
  tidx 		= threadIdx.x;
  ind       = count[iter]+blockIdx.x;
  
  while(ind<count[iter+1]){
      if (spkscore[ind]>ThS){
          
          tidy = threadIdx.y;          
          // only do this if the spike is "BAD"
          while (tidy<Nchan){
              X = 0.0f;              
              for (k=0;k<Nrank;k++)
                  X += W[tidx + id[ind]* nt0 + nt0*Nfilt*k] *
                          U[tidy + id[ind] * Nchan + Nchan*Nfilt*k];
                  
              dataraw[tidx + st[ind] + NT * tidy] += x[ind] * X;
              tidy += blockDim.y;
          }
      }
      ind += gridDim.x;
  }
}
//////////////////////////////////////////////////////////////////////////////////////////

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  /* Declare input variables*/
  double *Params, *d_Params;
  unsigned int nt0, Nchan, NT, Nfilt, Nnearest, Nrank, NchanU;

  
  /* read Params and copy to GPU */
  Params  	= (double*) mxGetData(prhs[0]);
  NT		= (unsigned int) Params[0];
  Nfilt     = (unsigned int) Params[1];
  nt0       = (unsigned int) Params[4];
  Nnearest  = (unsigned int) Params[5];
  Nrank     = (unsigned int) Params[6];
  NchanU    = (unsigned int) Params[10];  
  Nchan     = (unsigned int) Params[9];
  
  hipMalloc(&d_Params,      sizeof(double)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(double)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);

   /* collect input GPU variables*/
  mxGPUArray const  *W, *iList,  *U, *iC, *iW, *mu, *UtU, *wPCA;
  mxGPUArray *dWU, *draw, *nsp;  
  const int *d_iList, *d_iC, *d_iW;    
  const bool *d_UtU;    
  int *d_st,  *d_nsp, *d_ftype,  *d_id, *d_counter, *d_count;
  double *d_dWU;
  float *d_draw, *d_err, *d_x, *d_y, *d_z, *d_dout, *d_feat, *d_data,  *d_featPC, *d_eloss;  
  const float   *d_W, *d_U,  *d_mu, *d_wPCA;
  
  // draw is not a constant , so the data has to be "copied" over
  draw          = mxGPUCopyFromMxArray(prhs[1]);
  d_draw        = (float *)(mxGPUGetData(draw));
  U             = mxGPUCreateFromMxArray(prhs[2]);
  d_U        	= (float const *)(mxGPUGetDataReadOnly(U));
  W             = mxGPUCreateFromMxArray(prhs[3]);
  d_W        	= (float const *)(mxGPUGetDataReadOnly(W));
  mu            = mxGPUCreateFromMxArray(prhs[4]);
  d_mu          = (float const *)(mxGPUGetDataReadOnly(mu));
  iC            = mxGPUCreateFromMxArray(prhs[5]);
  d_iC          = (int const *)(mxGPUGetDataReadOnly(iC));
  iW            = mxGPUCreateFromMxArray(prhs[6]);
  d_iW        	= (int const *)(mxGPUGetDataReadOnly(iW));
  UtU           = mxGPUCreateFromMxArray(prhs[7]);
  d_UtU        	= (bool const *)(mxGPUGetDataReadOnly(UtU));
  iList         = mxGPUCreateFromMxArray(prhs[8]);
  d_iList       = (int const *)  (mxGPUGetDataReadOnly(iList));  
  wPCA          = mxGPUCreateFromMxArray(prhs[9]);
  d_wPCA        = (float const *)(mxGPUGetDataReadOnly(wPCA));
  
  
  const mwSize dimsNsp[] 	= {Nfilt,1};
  nsp 		= mxGPUCreateGPUArray(2, dimsNsp, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
  d_nsp 		= (int *)(mxGPUGetData(nsp));  
  const mwSize dimsdWU[] 	= {nt0, Nchan, Nfilt};
  dWU 		= mxGPUCreateGPUArray(3, dimsdWU, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
  d_dWU 		= (double *)(mxGPUGetData(dWU));
  
  hipMalloc(&d_dout,   2*NT * Nfilt* sizeof(float));
  hipMalloc(&d_data,   NT * Nfilt*Nrank* sizeof(float));
  
  hipMalloc(&d_err,   NT * sizeof(float));
  hipMalloc(&d_ftype, NT * sizeof(int));
  hipMalloc(&d_eloss,     NT * sizeof(float));
  hipMalloc(&d_st,    maxFR * sizeof(int));
  hipMalloc(&d_id,    maxFR * sizeof(int));
  hipMalloc(&d_x,     maxFR * sizeof(float));
  hipMalloc(&d_y,     maxFR * sizeof(float));
  hipMalloc(&d_z,     maxFR * sizeof(float));    
  
  hipMalloc(&d_counter,   2*sizeof(int));
  hipMalloc(&d_count,   nmaxiter*sizeof(int));
  hipMalloc(&d_feat,     maxFR * Nnearest * sizeof(float));
  hipMalloc(&d_featPC,     maxFR * NchanU*Nrank * sizeof(float));
  
  hipMemset(d_nsp,    0, Nfilt * sizeof(int));  
  hipMemset(d_dWU,    0, Nfilt * nt0 * Nchan* sizeof(double));
  hipMemset(d_dout,    0, NT * Nfilt * sizeof(float));
  hipMemset(d_data,    0, Nrank * NT * Nfilt * sizeof(float));
  hipMemset(d_counter, 0, 2*sizeof(int));
  hipMemset(d_count,    0, nmaxiter*sizeof(int));
  hipMemset(d_st,      0, maxFR *   sizeof(int));
  hipMemset(d_id,      0, maxFR *   sizeof(int));  
  hipMemset(d_x,       0, maxFR *    sizeof(float));
  hipMemset(d_y,       0, maxFR *    sizeof(float));
  hipMemset(d_z,       0, maxFR *    sizeof(float));
  hipMemset(d_feat,    0, maxFR * Nnearest *   sizeof(float));
  hipMemset(d_featPC,    0, maxFR * NchanU*Nrank *   sizeof(float));
  
  int *counter;
  counter = (int*) calloc(1,2 * sizeof(int));
  
  hipMemset(d_err,     0, NT * sizeof(float));
  hipMemset(d_ftype,   0, NT * sizeof(int));
  hipMemset(d_eloss,       0, NT *    sizeof(float));
  
  dim3 tpB(8, 2*nt0-1), tpF(16, Nnearest), tpS(nt0, 16), tpW(Nnearest, Nrank), tpPC(NchanU, Nrank);
  
  // filter the data with the spatial templates
  spaceFilter<<<Nfilt, Nthreads>>>(d_Params, d_draw, d_U, d_iC, d_iW, d_data);
  
  // filter the data with the temporal templates
  timeFilter<<<Nfilt, Nthreads>>>(d_Params, d_data, d_W, d_dout); 
  
  // compute the best filter
  bestFilter<<<NT/Nthreads,Nthreads>>>(d_Params, d_dout, d_mu, d_err, d_eloss, d_ftype);
  
  // loop to find and subtract spikes
  for(int k=0;k<(int) Params[3];k++){
      // ignore peaks that are smaller than another nearby peak
      cleanup_spikes<<<NT/Nthreads,Nthreads>>>(d_Params, d_dout, d_mu, d_err, d_eloss,
              d_ftype, d_st, d_id, d_x, d_y, d_z, d_counter);
      
      // add new spikes to 2nd counter
      hipMemcpy(counter, d_counter, 2*sizeof(int), hipMemcpyDeviceToHost);
      if (counter[0]>maxFR){
          counter[0] = maxFR;
          hipMemcpy(d_counter, counter, sizeof(int), hipMemcpyHostToDevice);
      }
      
      // extract template features before subtraction
      if (Params[12]>1)      
         extractFEAT<<<64, tpF>>>(d_Params, d_st, d_id, d_counter, d_dout, d_iList, d_mu, d_feat);
      
      // subtract spikes from raw data here
      subtract_spikes<<<Nfilt,tpS>>>(d_Params,  d_st, d_id, d_y, d_counter, d_draw, d_W, d_U);
  
      // filter the data with the spatial templates
       spaceFilterUpdate<<<Nfilt, 2*nt0-1>>>(d_Params, d_draw, d_U, d_UtU, d_iC, d_iW, d_data,
             d_st, d_id, d_counter);
      
       // filter the data with the temporal templates
       timeFilterUpdate<<<Nfilt, 2*nt0-1>>>(d_Params, d_data, d_W, d_UtU, d_dout,
               d_st, d_id, d_counter);

       if (counter[0]-counter[1]>0)
           bestFilterUpdate<<<counter[0]-counter[1], 2*nt0-1>>>(d_Params, d_dout, d_mu, 
                   d_err, d_eloss, d_ftype, d_st, d_id, d_counter);
       
      hipMemcpy(d_count+k+1, d_counter, sizeof(int), hipMemcpyDeviceToDevice);
       
      // update 1st counter from 2nd counter
      hipMemcpy(d_counter+1, d_counter, sizeof(int), hipMemcpyDeviceToDevice);
  }
  
  
  // compute PC features from reziduals + subtractions
  if (Params[12]>0)
     computePCfeatures<<<Nfilt, tpPC>>>(d_Params, d_counter, d_draw, d_st,
             d_id, d_y, d_W, d_U, d_mu, d_iW, d_iC, d_wPCA, d_featPC);

  //jic addition of time sorting prior to average_snips
  //get a set of indices for the sorted timestamp array
  //make a copy of the timestamp array to sort, plus an array of indicies

//   unsigned int *d_stSort, *d_idx;
//   hipMalloc(&d_stSort,  counter[0] * sizeof(int));
//   hipMemset(d_stSort, 0, counter[0] *sizeof(int));
//   hipMalloc(&d_idx,  counter[0] * sizeof(int));
//   hipMemset(d_idx, 0, counter[0] *sizeof(int));  
//   hipMemcpy( d_stSort, d_st, counter[0]*sizeof(int), hipMemcpyDeviceToDevice );  
//   set_idx<<< 1, 1 >>>(d_idx, counter[0]);
//   int left = 0;
//   int right = counter[0]-1;
//   cdp_simple_quicksort<<< 1, 1 >>>(d_stSort, d_idx, left, right, 0);

  // update dWU here by adding back to subbed spikes.
  // additional parameter d_idx = array of time sorted indicies  
  average_snips<<<Nfilt,tpS>>>(d_Params, d_st, d_id, d_x, d_y, d_counter, 
          d_draw, d_W, d_U, d_dWU, d_nsp,d_mu, d_z);
  
  float *x, *feat, *featPC, *vexp;
  int *st, *id;
  unsigned int minSize;
  if (counter[0]<maxFR)  minSize = counter[0];
  else                   minSize = maxFR;
  const mwSize dimst[] 	= {minSize,1}; 
  plhs[0] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  st = (int*) mxGetData(plhs[0]);
  plhs[1] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  id = (int*) mxGetData(plhs[1]);
  plhs[2] = mxCreateNumericArray(2, dimst, mxSINGLE_CLASS, mxREAL);
  x =  (float*) mxGetData(plhs[2]);  
  plhs[8] = mxCreateNumericArray(2, dimst, mxSINGLE_CLASS, mxREAL);
  vexp =  (float*) mxGetData(plhs[8]);  
  
  const mwSize dimsf[] 	= {Nnearest, minSize}; 
  plhs[3] = mxCreateNumericArray(2, dimsf, mxSINGLE_CLASS, mxREAL);
  feat =  (float*) mxGetData(plhs[3]);  
  
  // dWU stays a GPU array
  plhs[4] 	= mxGPUCreateMxArrayOnGPU(dWU);
  plhs[5] 	= mxGPUCreateMxArrayOnGPU(draw);
  plhs[6] 	= mxGPUCreateMxArrayOnGPU(nsp);    
      
  const mwSize dimsfPC[] 	= {NchanU, Nrank, minSize}; 
  plhs[7] = mxCreateNumericArray(3, dimsfPC, mxSINGLE_CLASS, mxREAL);
  featPC =  (float*) mxGetData(plhs[7]);  
  
  hipMemcpy(st, d_st, minSize * sizeof(int),   hipMemcpyDeviceToHost);
  hipMemcpy(id, d_id, minSize * sizeof(int),   hipMemcpyDeviceToHost);
  hipMemcpy(x,    d_y, minSize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vexp, d_x, minSize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(feat, d_feat, minSize * Nnearest*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(featPC,   d_featPC, minSize * NchanU*Nrank*sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(d_counter);
  hipFree(d_Params);
  hipFree(d_ftype);
  hipFree(d_err);
  hipFree(d_eloss);
  hipFree(d_st);
  hipFree(d_id);
  hipFree(d_x);  
  hipFree(d_y); 
  hipFree(d_z); 
  hipFree(d_feat);
  hipFree(d_featPC);
  hipFree(d_dout);
  hipFree(d_data);
//   hipFree(d_idx);
//   hipFree(d_stSort);

  mxGPUDestroyGPUArray(draw);  
  mxGPUDestroyGPUArray(wPCA);
  mxGPUDestroyGPUArray(dWU);
  mxGPUDestroyGPUArray(U);
  mxGPUDestroyGPUArray(UtU);
  mxGPUDestroyGPUArray(W);
  mxGPUDestroyGPUArray(mu);  
  mxGPUDestroyGPUArray(iC);  
  mxGPUDestroyGPUArray(nsp);
  mxGPUDestroyGPUArray(iW);  
  mxGPUDestroyGPUArray(iList);
  
}
